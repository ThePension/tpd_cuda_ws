#include "hip/hip_runtime.h"
#include "SliceGMHOST.h"

#include <assert.h>
#include <GM.h>
#include <GM_MemoryManagement.h>
#include <iostream>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void reductionIntraThreadGMHOST(float* tabGM,int nbSlice);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

SliceGMHOST::SliceGMHOST(Grid grid , int nbSlice , double* ptrPiHat , bool isVerbose) :
	RunnableGPU(grid, "SliceGM-HOST-" + to_string(nbSlice), isVerbose), // classe parente
//
	nbSlice(nbSlice), //
	ptrPiHat(ptrPiHat) //
    {
    // ntabGM
	{
	this->nTabGM = grid.threadCounts();
	}

    // MM
	{
	this->sizeTabGM = nTabGM * sizeof(float);
	GM::malloc(&tabGM, sizeTabGM);
	}
    }

SliceGMHOST::~SliceGMHOST(void)
    {
    //MM (device free)
	{
	GM::free(tabGM);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * <pre>
 * Idea globale
 *
 *	Etape 0 : Promotion d'un tableau en GM (MemoryManagement MM)		(Dans le constructeur)
 * 	Etape 1 : Reduction intra-thread dans un tableau promu en GM
 * 	Etape 2 : Copy du tableau coter host
 * 	Etape 3 : Reduction  du tableau coter host
 * 	Etape 4 : Destruction GM						(Dans le destructeur)
 *
 * </pre>
 */
void SliceGMHOST::run()
    {
    reductionIntraThreadGMHOST<<<dg, db>>>(tabGM, nTabGM);
    reductionHost();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Reduction paralle sur cpu du tableau promu en GM, ramener coter host
 */
void SliceGMHOST::reductionHost()
    {
    float* tab = new float[sizeTabGM];
    GM::memcpyDToH(tab, tabGM, sizeTabGM);

    float sum = 0.f;

//#pragma omp parallel for reduction(+:sum)
    for (size_t i = 0; i < nTabGM; i++)
	{
	sum += tab[i];
	}

    *ptrPiHat = sum / nTabGM;
    delete[] tab;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
