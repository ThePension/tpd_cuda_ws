#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Thread2D.cu.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * Chaque thread effecteur une reduction intrathread avec le patern d'entrelacement,
 * puis stocke son resultat dans SA case dans tabGM
 *
 * tabGM est un tableau promu, qui a autant de case que de thread
 * </pre>
 */
__global__ void reductionIntraThreadGMHOST(float* tabGM , int nbSlice)
    {
    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();
    const float DX = 1.f / nbSlice;

    int s = TID;
    float sum = 0.f;
    while (s < nbSlice)
	{
	sum += f(s * DX);
	s += NB_THREAD;
	}

    tabGM[TID] = sum;

    // Conseils :
    //
    //		(C1) 	Ne calculer pas en double cote device, mais tout en float.
    //			En particulier, on ecrira 4.0f et non 4 (meme si ici le compilateur va l'optimiser a notre place, mais c'est bien de la faire par principe)
    //
    // 		(C2) 	Effectuez plutot le fois DX de l'aire du slice une seule fois par Thread, que pour chaque slice,
    //          	 ou qu'une seule fois cote host (debordement de type float cote device, car on ne fait que sommer?)
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    return 4.f / (1.f + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

