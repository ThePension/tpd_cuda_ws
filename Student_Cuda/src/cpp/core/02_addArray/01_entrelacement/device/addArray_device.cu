#include "hip/hip_runtime.h"
#include "Thread2D.cu.h"
#include "Thread1D.cu.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * output : void required, car kernel is asynchrone
 * pattern entrelacement
 */
__global__ void addArray(float* ptrDevV1 , float* ptrDevV2 , float* ptrDevW , int n)
    {
    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();

    // Debug, facultatif
//    if (TID == 0)
//	{
//	printf("Coucou from device tid = %d", TID);
//	}

    // TODO addArray
    // Je suis un chinois, qu'est-ce que je dois faire ?
    int s = TID;

    while(s < n)
	{
	ptrDevW[s] += ptrDevV1[s] + ptrDevV2[s];
	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

