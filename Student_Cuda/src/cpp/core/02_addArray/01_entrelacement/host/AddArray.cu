#include "hip/hip_runtime.h"
#include "AddArray.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "VectorTools.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addArray(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddArray::AddArray(const Grid& grid , float* ptrV1 , float* ptrV2 , float* ptrW , int n) :
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n), //
	dg(grid.dg), //
	db(grid.db)
    {
    this->sizeVector = n * sizeof(float);

    // MM (malloc Device)
	{
	GM::malloc(&ptrDevV1, sizeVector);
	GM::malloc(&ptrDevV2, sizeVector);
	GM::malloc(&ptrDevW, sizeVector);
	}

    }

AddArray::~AddArray(void)
    {
    //MM (device free)
	{
	GM::free(ptrDevV1);
	GM::free(ptrDevV2);
	GM::free(ptrDevW);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * override
 */
void AddArray::run()
    {
    // MM (copy Host->Device)
	{
	GM::memcpyHToD(ptrDevV1, ptrV1, sizeVector);
	GM::memcpyHToD(ptrDevV2, ptrV2, sizeVector);
	}

	addArray<<<dg, db>>>(ptrDevV1, ptrDevV2, ptrDevW, n);

    //Kernel::synchronize();// inutile

    // MM (Device -> Host)
	{
	GM::memcpyDToH(ptrW, ptrDevW, sizeVector);
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
