#include "hip/hip_runtime.h"
#include "AddArray11.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "VectorTools.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addArray11(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddArray11::AddArray11(const Grid& grid , float* ptrV1 , float* ptrV2 , float* ptrW , int n) :
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n), //
	dg(grid.dg), //
	db(grid.db)
    {
    this->sizeVector = -1; // TODO addArray11 // octet

    // MM (malloc Device)
	{
	// TODO addArray11
	}
    }

AddArray11::~AddArray11(void)
    {
    //MM (device free)
	{
	// TODO addArray11
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * override
 */
void AddArray11::run()
    {
    // MM (copy Host->Device)
	{
	// TODO addArray11
	}

    assert(dg.x * dg.y * dg.z * db.x * db.y * db.z == n);
    // TODO addArray11 call kernel

    // MM (Device -> Host)
	{
	// TODO addArray11
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
